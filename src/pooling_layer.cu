/*
 * \file pooling_layer.cu
 */

#include "pooling_layer.h"

void Pooling::InitFeatureShape() {
    out_shape_[0] = in_shape_[0];
    out_shape_[1] = in_shape_[1];
    out_shape_[2] = (in_shape_[2] + 2 * padding_ - kernel_size_) / stride_ + 1;
    out_shape_[3] = (in_shape_[3] + 2 * padding_ - kernel_size_) / stride_ + 1;
}

void Pooling::InitWeightsShape(std::vector<std::array<int, 4>> &w_l,
                               std::vector<std::array<int, 4>> &b_l) {
    w_l.emplace_back(std::array<int, 4>{0, 0, 0, 0});
    b_l.emplace_back(std::array<int, 4>{0, 0, 0, 0});
    return;
}

void Pooling::Forward() {
    input_desc_ = input_.tensor();
    output_desc_ = output_.tensor();
    hipdnnPoolingForward(cuda_->cudnn(), pool_desc_, &cuda_->one, input_desc_,
                        input_.CudaPtr(), &cuda_->zero, output_desc_,
                        d_retain_output_);
    checkCudaErrors(hipMemcpy(output_.CudaPtr(), d_retain_output_,
                               output_.buf_size(), hipMemcpyDeviceToDevice));
#if (DEBUG_POOLING & 0x01)
    input_.print(name_ + "::input", true, input_.GetWidth());
    output_.print(name_ + "::output", true, output_.GetWidth());
#endif

    return;
}

void Pooling::Backward(BlobPointer<float> const &labels) {
    checkCudnnErrors(hipdnnPoolingBackward(
        cuda_->cudnn(), pool_desc_, &cuda_->one, output_desc_, d_retain_output_,
        output_desc_, output_.CudaPtr(), input_desc_, input_.CudaPtr(),
        &cuda_->zero, input_desc_, d_temp_grad_features_));
    this->BackwardCopy();

#if (DEBUG_POOLING & 0x02)
    std::cout << name_ << "[BACKWARD]" << std::endl;
    input_.print(name_ + "::input", true, input_.GetWidth());
    output_.print(name_ + "::predict", true, output_.GetWidth());
    grad_output_.print(name_ + "::dy", true, grad_output_.GetWidth());
    grad_input_.print(name_ + "::dx", true, grad_input_.GetWidth());
#endif
    return;
}
