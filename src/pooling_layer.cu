/*
 * \file pooling_layer.cu
 */

#include "pooling_layer.h"

std::array<int, 4> Pooling::InitFeatureShape(std::array<int, 4> const &in_shape)
{

	in_shape_ = in_shape;

	out_shape_[0] = in_shape_[0];
	out_shape_[1] = in_shape_[1];
	out_shape_[2] = (in_shape_[2] + 2 * padding_ - kernel_size_) / stride_ + 1;
	out_shape_[3] = (in_shape_[3] + 2 * padding_ - kernel_size_) / stride_ + 1;

	return out_shape_;
}

void Pooling::InitWeightsShape(std::vector<std::array<int, 4>> &w_l, std::vector<std::array<int, 4>> &b_l)
{

	w_l.emplace_back(std::array<int, 4>{0, 0, 0, 0});
	b_l.emplace_back(std::array<int, 4>{0, 0, 0, 0});
	return;
}

void Pooling::Forward()
{

	input_desc_ = input_.tensor();
	output_desc_ = output_.tensor();
	hipdnnPoolingForward(cuda_->cudnn(), pool_desc_,
						&cuda_->one, input_desc_, input_.CudaPtr(),
						&cuda_->zero, output_desc_, output_.CudaPtr());
#if (DEBUG_POOLING & 0x01)
	input_.print(name_ + "::input", true, input_.GetWidth());
	output_.print(name_ + "::output", true, output_.GetWidth());
#endif

	return;
}

void Pooling::Backward(BlobPointer<flt_type> const &labels)
{

	checkCudnnErrors(
		hipdnnPoolingBackward(cuda_->cudnn(), pool_desc_,
							 &cuda_->one,
							 output_desc_, output_.CudaPtr(),
							 output_desc_, grad_output_.CudaPtr(),
							 input_desc_, input_.CudaPtr(),
							 &cuda_->zero,
							 input_desc_, grad_input_.CudaPtr()));

#if (DEBUG_POOLING & 0x02)
	std::cout << name_ << "[BACKWARD]" << std::endl;
	input_.print(name_ + "::input", true, input_.GetWidth());
	output_.print(name_ + "::predict", true, output_.GetWidth());
	grad_output_.print(name_ + "::dy", true, grad_output_.GetWidth());
	grad_input_.print(name_ + "::dx", true, grad_input_.GetWidth());
#endif
	return;
}
