#include "hip/hip_runtime.h"
/*
 * \file fully_connected_layer.cu
 */

#include "fully_connected_layer.h"

__global__ void InitiateVecOnes(flt_type *d_one_vec, size_t length)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= length)
        return;

    d_one_vec[i] = 1.f;
}

std::array<int, 4> Fully_connected::InitFeatureShape(std::array<int, 4> const &in_shape)
{

    in_shape_[0] = in_shape[0];
    in_shape_[1] = in_shape[1] * in_shape[2] * in_shape[3];
    in_shape_[2] = 1;
    in_shape_[3] = 1;

    out_shape_[0] = in_shape[0];
    out_shape_[1] = output_shape_;
    out_shape_[2] = 1;
    out_shape_[3] = 1;

    return out_shape_;
}

void Fully_connected::InitWeightsShape(std::vector<std::array<int, 4>> &w_l, std::vector<std::array<int, 4>> &b_l)
{
    // initialize weight, bias, and output
    input_shape_ = in_shape_[1];

    std::array<int, 4> w{1, 1, input_shape_, output_shape_}, b{1, 1, output_shape_, 1};
    w_l.emplace_back(w);
    b_l.emplace_back(b);

    return;
}

void Fully_connected::DescriptorsAndWorkSpace()
{

    if (d_one_vec != nullptr)
    {
        checkCudaErrors(hipFree(d_one_vec));
    }

    int batch_size = input_.get_n();
    checkCudaErrors(hipMalloc((void **)&d_one_vec, sizeof(flt_type) * batch_size));
    InitiateVecOnes<<<(batch_size + BLOCK_DIM_1D - 1) / BLOCK_DIM_1D, BLOCK_DIM_1D>>>(d_one_vec, batch_size);
}

void Fully_connected::Forward()
{

    int batch_size = input_.get_n();
    // output = weightsT * input (without biases)
    checkCublasErrors(
        hipblasSgemm(cuda_->cublas(),
                    HIPBLAS_OP_T, HIPBLAS_OP_N,
                    output_shape_, batch_size, input_shape_,
                    &cuda_->one,
                    weights_.CudaPtr(), input_shape_,
                    input_.CudaPtr(), input_shape_,
                    &cuda_->zero,
                    output_.CudaPtr(), output_shape_));

    // output += biases * d-one-vecT
    checkCublasErrors(
        hipblasSgemm(cuda_->cublas(),
                    HIPBLAS_OP_N, HIPBLAS_OP_N,
                    output_shape_, batch_size, 1,
                    &cuda_->one,
                    biases_.CudaPtr(), output_shape_,
                    d_one_vec, 1,
                    &cuda_->one,
                    output_.CudaPtr(), output_shape_));

#if (DEBUG_DENSE & 0x01)
    input_.print(name_ + "::input", true);
    weights_.print(name_ + "::weight", true);
    biases_.print(name_ + "::bias", true);
    output_.print(name_ + "::output", true);
#endif
    return;
}

void Fully_connected::Backward(BlobPointer<flt_type> const &labels)
{

    int batch_size = input_.get_n();
    hipblasSgemv(cuda_->cublas(),
                HIPBLAS_OP_N,
                output_shape_, batch_size,
                &cuda_->one,
                grad_output_.CudaPtr(), output_shape_,
                d_one_vec, 1,
                &cuda_->zero,
                grad_biases_.CudaPtr(), 1);

    // dw = x * (dy)T
    hipblasSgemm(cuda_->cublas(),
                HIPBLAS_OP_N, HIPBLAS_OP_T,
                input_shape_, output_shape_, batch_size,
                &cuda_->one,
                input_.CudaPtr(), input_shape_,
                grad_output_.CudaPtr(), output_shape_,
                &cuda_->zero,
                grad_weights_.CudaPtr(), input_shape_);

    // dx = W * dy
    if (!gradient_stop_)
        hipblasSgemm(cuda_->cublas(),
                    HIPBLAS_OP_N, HIPBLAS_OP_N,
                    input_shape_, batch_size, output_shape_,
                    &cuda_->one,
                    weights_.CudaPtr(), input_shape_,
                    grad_output_.CudaPtr(), output_shape_,
                    &cuda_->zero,
                    grad_input_.CudaPtr(), input_shape_);

#if (DEBUG_DENSE & 0x02)
    std::cout << name_ << "[BACKWARD]" << std::endl;
    grad_output_.print(name_ + "::gradients", true);
    grad_weights_.print(name_ + "::gfilter", true);
    grad_biases_.print(name_ + "::gbias", true);
    grad_input_.print(name_ + "::gdata", true);
#endif

    return;
}
