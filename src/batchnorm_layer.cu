/*
 * \file convolutional_layer.cu
 */

#include <iostream>

#include "batchnorm_layer.h"

void Batchnorm2D::InitiateWeightsAndBiases() {
    if (weights_.CudaPtr() == nullptr || biases_.CudaPtr() == nullptr) return;

    std::vector<float> weights(weights_.LengthNchw(), 1.0);
    weights_.ToDevice(weights.data(), weights.size());

    std::vector<float> biases(biases_.LengthNchw(), 0.0);
    for (size_t i = 0; i < biases.size(); i++) biases[i] = 0.f;
    biases_.ToDevice(biases.data(), biases.size());
}

void Batchnorm2D::AllocateBatchnorm2D() {
    if (resultRunningMean_ == nullptr) {
        checkCudaErrors(hipMalloc((void **)&resultRunningMean_,
                                   sizeof(float) * input_.GetChannels()));
        checkCudaErrors(hipMemset(resultRunningMean_, 0,
                                   sizeof(float) * input_.GetChannels()));
    }

    if (resultRunningVariance_ == nullptr) {
        checkCudaErrors(hipMalloc((void **)&resultRunningVariance_,
                                   sizeof(float) * input_.GetChannels()));
        checkCudaErrors(hipMemset(resultRunningVariance_, 0,
                                   sizeof(float) * input_.GetChannels()));
    }

    if (resultSaveMean_ == nullptr) {
        checkCudaErrors(hipMalloc((void **)&resultSaveMean_,
                                   sizeof(float) * input_.GetChannels()));
    }

    if (resultSaveInvVariance_ == nullptr) {
        checkCudaErrors(hipMalloc((void **)&resultSaveInvVariance_,
                                   sizeof(float) * input_.GetChannels()));
    }
}

std::array<int, 4> Batchnorm2D::InitFeatureShape(
    std::array<int, 4> const &in_shape) {
    out_shape_ = in_shape_ = in_shape;
    return out_shape_;
}

void Batchnorm2D::InitWeightsShape(std::vector<std::array<int, 4>> &w_l,
                                   std::vector<std::array<int, 4>> &b_l) {
    // 1xCx1x1
    w_l.emplace_back(std::array<int, 4>{1, in_shape_[1], 1, 1});
    b_l.emplace_back(std::array<int, 4>{1, in_shape_[1], 1, 1});
    return;
}

void Batchnorm2D::DescriptorsAndWorkSpace() {
    output_desc_ = output_.tensor();
    input_desc_ = input_.tensor();

    hipdnnDeriveBNTensorDescriptor(bnDesc_, input_desc_,
                                  HIPDNN_BATCHNORM_SPATIAL);
    AllocateBatchnorm2D();
    return;
}

void Batchnorm2D::Forward() {
    if (phase_ == WorkloadType::training) {
        checkCudnnErrors(hipdnnBatchNormalizationForwardTraining(
            cuda_->cudnn(),  // hipdnnHandle_t                    handle,
            HIPDNN_BATCHNORM_SPATIAL,  // hipdnnBatchNormMode_t             mode,
            &cuda_->one,              // const void                      *alpha,
            &cuda_->zero,             // const void                      *beta,
            input_desc_,              // const hipdnnTensorDescriptor_t    xDesc,
            input_.CudaPtr(),         // const void                      *x,
            output_desc_,             // const hipdnnTensorDescriptor_t    yDesc,
            output_.CudaPtr(),        // void                            *y,
            bnDesc_,                  // const hipdnnTensorDescriptor_t
                                      // bnScaleBiasMeanVarDesc,
            weights_.CudaPtr(),  // const void                      *bnScale,
            biases_.CudaPtr(),   // const void                      *bnBias,
            exponentialAverageFactor_,  // double exponentialAverageFactor,
            resultRunningMean_,         // void *resultRunningMean,
            resultRunningVariance_,     // void *resultRunningVariance,
            epsilon_,               // double                           epsilon,
            resultSaveMean_,        // void                    *resultSaveMean,
            resultSaveInvVariance_  // void *resultSaveInvVariance
            ));
    } else {
        checkCudnnErrors(hipdnnBatchNormalizationForwardInference(
            cuda_->cudnn(),  // hipdnnHandle_t                    handle,
            HIPDNN_BATCHNORM_SPATIAL,  // hipdnnBatchNormMode_t             mode,
            &cuda_->one,              // const void                      *alpha,
            &cuda_->zero,             // const void                      *beta,
            input_desc_,              // const hipdnnTensorDescriptor_t    xDesc,
            input_.CudaPtr(),         // const void                      *x,
            output_desc_,             // const hipdnnTensorDescriptor_t    yDesc,
            output_.CudaPtr(),        // void                            *y,
            bnDesc_,                  // const hipdnnTensorDescriptor_t
                                      // bnScaleBiasMeanVarDesc,
            weights_.CudaPtr(),  // const void                      *bnScale,
            biases_.CudaPtr(),   // const void                      *bnBias,
            resultRunningMean_,  // const void                *estimatedMean,
            resultRunningVariance_,  // const void *estimatedVariance,
            epsilon_                 // double                           epsilon
            ));
    }

    return;
}

void Batchnorm2D::Backward(BlobPointer<float> const &labels) {
    checkCudnnErrors(hipdnnBatchNormalizationBackward(
        cuda_->cudnn(),           // hipdnnHandle_t                    handle,
        HIPDNN_BATCHNORM_SPATIAL,  // hipdnnBatchNormMode_t             mode,
        &cuda_->one,              // const void                 *alphaDataDiff,
        &cuda_->zero,             // const void                 *betaDataDiff,
        &cuda_->one,              // const void                 *alphaParamDiff,
        &cuda_->zero,             // const void                 *betaParamDiff,
        input_desc_,              // const hipdnnTensorDescriptor_t    xDesc,
        input_.CudaPtr(),         // const void                      *x,
        output_desc_,             // const hipdnnTensorDescriptor_t    dyDesc,
        grad_output_.CudaPtr(),   // const void                      *dy,
        input_desc_,              // const hipdnnTensorDescriptor_t    dxDesc,
        grad_input_.CudaPtr(),    // void                            *dx,
        bnDesc_,                  // const hipdnnTensorDescriptor_t
                                  // bnScaleBiasDiffDesc,
        weights_.CudaPtr(),       // const void                      *bnScale,
        grad_weights_.CudaPtr(),  // void *resultBnScaleDiff,
        grad_biases_.CudaPtr(),   // void *resultBnBiasDiff,
        epsilon_,                 // double                           epsilon,
        resultSaveMean_,          // const void                      *savedMean,
        resultSaveInvVariance_    // const void *savedInvVariance
        ));
    return;
}
