/*
 * \file convolutional_layer.cu
 */

#include <iostream>

#include "residual_layer.h"

std::array<int, 4> Residual::InitFeatureShape(
    std::array<int, 4> const &in_shape) {
    out_shape_ = in_shape_ = in_shape;
    return out_shape_;
}

void Residual::InitWeightsShape(std::vector<std::array<int, 4>> &w_l,
                                std::vector<std::array<int, 4>> &b_l) {
    // nothing actually
    w_l.emplace_back(std::array<int, 4>{1, 1, 1, 1});
    b_l.emplace_back(std::array<int, 4>{1, 1, 1, 1});
    return;
}

void Residual::DescriptorsAndWorkSpace() {
    output_desc_ = output_.tensor();
    input_desc_ = input_.tensor();
    return;
}

void Residual::Forward() {
    checkCudaErrors(
        hipMemset(output_.CudaPtr(), 0, sizeof(float) * output_.LengthNchw()));
    hipdnnAddTensor(
        cuda_->cudnn(),  // hipdnnHandle_t                     handle,
        &cuda_->one,     // const void                       *alpha,
        inputLayer1_->GetOutputDesc(),  // const hipdnnTensorDescriptor_t aDesc,
        inputLayer1_->GetOutput().CudaPtr(),  // const void *A,
        &cuda_->zero,      // const void                       *beta,
        output_desc_,      // const hipdnnTensorDescriptor_t     cDesc,
        output_.CudaPtr()  // void                             *C
    );
    hipdnnAddTensor(
        cuda_->cudnn(),  // hipdnnHandle_t                     handle,
        &cuda_->one,     // const void                       *alpha,
        inputLayer2_->GetOutputDesc(),  // const hipdnnTensorDescriptor_t aDesc,
        inputLayer2_->GetOutput().CudaPtr(),  // const void *A,
        &cuda_->zero,      // const void                       *beta,
        output_desc_,      // const hipdnnTensorDescriptor_t     cDesc,
        output_.CudaPtr()  // void                             *C
    );
    return;
}

void Residual::Backward(BlobPointer<float> const &labels) {
    checkCudaErrors(hipMemcpy(
        inputLayer1_->GetGradOutput().CudaPtr(), grad_output_.CudaPtr(),
        sizeof(float) * output_.LengthNchw(), hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(
        inputLayer2_->GetGradOutput().CudaPtr(), grad_output_.CudaPtr(),
        sizeof(float) * output_.LengthNchw(), hipMemcpyDeviceToDevice));
    return;
}
