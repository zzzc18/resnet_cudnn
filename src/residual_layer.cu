/*
 * \file convolutional_layer.cu
 */

#include <iostream>

#include "residual_layer.h"

void Residual::InitFeatureShape() { out_shape_ = in_shape_; }

void Residual::InitWeightsShape(std::vector<std::array<int, 4>> &w_l,
                                std::vector<std::array<int, 4>> &b_l) {
    // nothing actually
    w_l.emplace_back(std::array<int, 4>{1, 1, 1, 1});
    b_l.emplace_back(std::array<int, 4>{1, 1, 1, 1});
    return;
}

void Residual::DescriptorsAndWorkSpace() {
    output_desc_ = output_.tensor();
    input_desc_ = input_.tensor();
    return;
}

void Residual::Forward() {
    InitiateZeros<<<(output_.LengthNchw() + BLOCK_DIM_1D - 1) / BLOCK_DIM_1D,
                    BLOCK_DIM_1D>>>(output_.CudaPtr(), output_.LengthNchw());
    // checkCudaErrors(
    //     hipMemset(output_.CudaPtr(), 0, sizeof(float) *
    //     output_.LengthNchw()));
    hipdnnAddTensor(
        cuda_->cudnn(),  // hipdnnHandle_t                     handle,
        &cuda_->one,     // const void                       *alpha,
        inputLayer1_->GetOutputDesc(),  // const hipdnnTensorDescriptor_t aDesc,
        inputLayer1_->GetOutput().CudaPtr(),  // const void *A,
        &cuda_->one,       // const void                       *beta,
        output_desc_,      // const hipdnnTensorDescriptor_t     cDesc,
        output_.CudaPtr()  // void                             *C
    );
    hipdnnAddTensor(
        cuda_->cudnn(),  // hipdnnHandle_t                     handle,
        &cuda_->one,     // const void                       *alpha,
        inputLayer2_->GetOutputDesc(),  // const hipdnnTensorDescriptor_t aDesc,
        inputLayer2_->GetOutput().CudaPtr(),  // const void *A,
        &cuda_->one,       // const void                       *beta,
        output_desc_,      // const hipdnnTensorDescriptor_t     cDesc,
        output_.CudaPtr()  // void                             *C
    );
    return;
}

void Residual::Backward(BlobPointer<float> const &labels) {
    checkCudaErrors(hipMemcpy(
        inputLayer1_->GetGradOutput().CudaPtr(), grad_output_.CudaPtr(),
        sizeof(float) * output_.LengthNchw(), hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(
        inputLayer2_->GetGradOutput().CudaPtr(), grad_output_.CudaPtr(),
        sizeof(float) * output_.LengthNchw(), hipMemcpyDeviceToDevice));
    return;
}
